
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include "hiprand/hiprand_kernel.h"


const int THREADS_PER_BLOCK       = 32; //2**5
const int BLOCKS                  = 128; //2**7
const int THREADS_EVER            = THREADS_PER_BLOCK * BLOCKS ;//2**12

//const int STARS                   = 1024 * 1024 * 64 ;//2**26
const int STARS                   = 1024 * 1024 * 4;//2**16
//const int STARS                   = THREADS_EVER * 2;
const int NEIGHBORHOODS           = THREADS_EVER ;
const int NEIGHBORHOOD_STARS      = STARS / NEIGHBORHOODS ;//2**14

const int ITERATIONS              = 1000 ;//at 1m years / iteration
// when debugging this is the minimal
// const int ITERATIONS              = 2 ;//at 1m years / iteration

typedef uint8_t output_t ;
const char * OUTPUT_T_FORMAT = "%u" ;

/*
 * see notes-20121226.md for description of the semantics of the sim
 */

/*
// the code I want
enum class States : output_t { 
  PROTOSTAR = 0,
  UNINHABITABLE = 1,
  INHABITABLE = 2,
  CELLULAR = 3, 
  OXYGEN_EVENT = 4,
  CAMBRIAN_EVENT = 5,
  TECHNICAL_CIV = 6
  //SPACE_FARING = 7
};
*/
// the code I have: nvcc does not support -std=c++0x.
// I am refusing to deal with pre c++11 enum.
const output_t  PROTOSTAR = 0;
const output_t  UNINHABITABLE = 1;
const output_t  INHABITABLE = 2;
const output_t  CELLULAR = 3;
const output_t  OXYGEN_EVENT = 4;
const output_t  CAMBRIAN_EVENT = 5;
const output_t  TECHNICAL_CIV = 6;
const output_t  SPACE_FARING = 7;

const unsigned int  NUM_STATES = 8;

// **********************************************************************
// each step is a boolean trial representing 10myr.
//
// the steps are indexed by their state values.
//
// at each step we generate a flat random 0.0f to 1.0f.
//
// Then we convert to 0 if less than the transition probability or 1
// if more than transition probability.  This means that 0 is a transition
// and 1 is no change. 
//
// note civ states (eg: spacefaring, collapse, colonize) are modeled 
// separately in fast time since 10myr steps don't seem reasonable
//
// STATE_CHANGES is indexed by pseudo code
//   flip{0,1} * NUM_STATES + current_state{0..(NUM_STATES -1)}
//
//   where flip=0 implies transistion and 1 implies stasis and
//   where the value returned by that is the next state
//**********************************************************************

const float pZ = 0.0 ;   //uninhabitable
const float pH = 0.5 ;   //habitable
const float pB = 0.007 ; //Cellular Biology (50% chance after 100*10myr )
const float pO = 0.007 ; //Oxygen Event (took another 100 iterations on earth)
const float pM = 0.0035; //Cambrian Explosion of [M]ulticellular life (took 200 iterations on earth)
const float pT = 0.014;  //Technical Civilization (radio telescope)
const float pS = 0.5  ;  //Space Faring Civilization (or collapse) per 1000 years of Tech Civ
const float pC = 0.5  ;  //spacefaring local solar system civ collapse

const output_t STATE_CHANGES [ NUM_STATES * 2 ] =
{
  UNINHABITABLE,  INHABITABLE,      //0,1 states from PROTOSTAR,
  UNINHABITABLE,  UNINHABITABLE,    //0,1 states from UNINHABITABLE
  INHABITABLE,    CELLULAR,         //0,1 states from INHABITABLE
  CELLULAR,       OXYGEN_EVENT,     //0,1 states from CELLULAR
  OXYGEN_EVENT,   CAMBRIAN_EVENT,   //0,1 states from OXYGEN_EVENT
  CAMBRIAN_EVENT, TECHNICAL_CIV,    //0,1 states from CAMBRIAN_EVENT
  CAMBRIAN_EVENT, SPACE_FARING,     //0,1 states from TECHNICAL_CIV
  SPACE_FARING,   TECHNICAL_CIV,    //0,1 states from SPACE_FARING
};

const float P_CHANGE [ NUM_STATES ] =
{
  pH,  //p(change) from PROTOSTAR,
  pZ,  //p(change) from UNINHABITABLE
  pB,  //p(change) from INHABITABLE
  pO,  //p(change) from CELLULAR
  pM,  //p(change) from OXYGEN_EVENT
  pT,  //p(change) from CAMBRIAN_EVENT
  pS,  //p(change) from TECHNICAL_CIV
  pC,  //p(change) from SPACE_FARING
};


__global__ void
init_rands(unsigned int seed, hiprandStateXORWOW_t *rgens )
{
  int baseIdx = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init( seed, baseIdx, 0, & rgens[baseIdx]);
  //printf( "init_rands %i %u thr=%i blck=%i blckdim=%i\n", baseIdx, seed, threadIdx.x, blockIdx.x, blockDim.x );
}

__global__ void
init_buf( output_t *outs, int n)
{
  int baseIdx = threadIdx.x + blockIdx.x * blockDim.x;
  for( int i=0; i<n; ++i ) {
    outs[baseIdx * n + i] = PROTOSTAR;
  }
}

__global__ void
iterate_states(
    hiprandStateXORWOW_t *rgens,
    output_t *buf_in,
    output_t *buf_out,
    int neighborhood_stars,
    output_t *state_matrix,
    float *pchange//,
    //output_t *buf_old_states,
    //float *buf_rnds,
    //unsigned int *_flips
    )
{
  int neighborhood = threadIdx.x + blockIdx.x * blockDim.x;
  int base = neighborhood * neighborhood_stars;
  hiprandStateXORWOW_t rgen = rgens[neighborhood];
  for( int i=0; i<neighborhood_stars; ++i ) {
    const int star = base + i;
    output_t old_state = buf_in[star];

    //float rnd = curand_uniform(&rgen);
    //float p = pchange[ old_state ];
    //unsigned int flip = 1 - (unsigned int) ceil( rnd - p );
    //unsigned int new_state_index = old_state * 2 + flip ;
    //output_t new_state = state_matrix[ new_state_index ];
    //buf_out[star] = new_state;
    //buf_out[star] = i % NUM_STATES;

    //printf("neighborhood=%i neighborhood_stars=%i i=%i star=%i old_state=%i rnd=%f p=%f flip=%i new_state_index=%i new_state=%i\n",
    //  neighborhood,
    //  neighborhood_stars,
    //  i,
    //  star,
    //  old_state,
    //  rnd,
    //  p,
    //  flip,
    //  new_state_index,
    //  new_state
    //  );
    //
    // the the inner loop was before I hacked in the debug version above
    //
    buf_out[star] = state_matrix[
        old_state * 2 +
        (1 -
          (unsigned int) ceil(hiprand_uniform(&rgen) -
          pchange[ old_state ]))
      ];
  }
  rgens[neighborhood] = rgen;
}

unsigned int
generate_seed()
{
  FILE* randomSource = fopen("/dev/random", "rb");
  unsigned int seed;
  int recordsRead = fread( &seed, sizeof(unsigned int), 1, randomSource );
  fclose( randomSource );
  return seed;
}

#define STAR_STATE_CHAR(x) (int)((output_t)('0') + x)

void
inspect( const char prefix, const output_t *array, const int line, const int n )
{
  for( int i=0; i<n; ++i ) {
    if( i % line == 0 ){
      putchar(prefix);
    }
    putchar(STAR_STATE_CHAR(array[i]));
    if( (i + 1) % line == 0 ){
      putchar('\n');
    }
  }
}

int
main()
{
  output_t *outs ;
  output_t *couts1;
  output_t *couts2;
  output_t *cstate_matrix;
  output_t *cstate_matrix_shadow;
  float *cpchange;
  hiprandStateXORWOW_t *crgens;

  unsigned int seed = generate_seed();

  const int output_size = STARS * sizeof( output_t );
  const int rgen_size =   NEIGHBORHOODS * sizeof( hiprandStateXORWOW_t );
  const int state_matrix_size = 2 * NUM_STATES * sizeof( output_t );
  const int pchange_size = NUM_STATES * sizeof( float );

  hipMalloc( (void**)&couts1, output_size );
  hipMalloc( (void**)&couts2, output_size );
  hipMalloc( (void**)&cstate_matrix, state_matrix_size );
  hipMalloc( (void**)&cpchange, pchange_size );
  hipMalloc( (void**)&crgens, rgen_size );


  //TODO: are these being copied to the host correctly?
  hipMemcpy( cstate_matrix, STATE_CHANGES, state_matrix_size, hipMemcpyHostToDevice );
  hipMemcpy( cpchange, P_CHANGE, pchange_size, hipMemcpyHostToDevice );


  init_buf<<<BLOCKS, THREADS_PER_BLOCK>>>( couts1, NEIGHBORHOOD_STARS );
  init_buf<<<BLOCKS, THREADS_PER_BLOCK>>>( couts2, NEIGHBORHOOD_STARS );
  init_rands<<<BLOCKS, THREADS_PER_BLOCK>>>( seed, crgens );


  hipHostAlloc(&cstate_matrix_shadow, state_matrix_size, sizeof(output_t));
  hipMemcpy( cstate_matrix_shadow, cstate_matrix, state_matrix_size, hipMemcpyDeviceToHost );
  inspect( 'S', STATE_CHANGES, 2, state_matrix_size);
  inspect( 's', cstate_matrix_shadow, 2, state_matrix_size);

  hipHostAlloc(&outs, STARS, sizeof(output_t));
  hipMemcpy( outs, couts1, output_size, hipMemcpyDeviceToHost );
  inspect( 'x', outs, NEIGHBORHOOD_STARS, STARS );
  hipMemcpy( outs, couts2, output_size, hipMemcpyDeviceToHost );
  inspect( 'X', outs, NEIGHBORHOOD_STARS, STARS );

  for( int i=0; i< ITERATIONS; i += 2 ){
    iterate_states<<<BLOCKS, THREADS_PER_BLOCK>>>( crgens, couts1, couts2, NEIGHBORHOOD_STARS, cstate_matrix, cpchange );
    iterate_states<<<BLOCKS, THREADS_PER_BLOCK>>>( crgens, couts2, couts1, NEIGHBORHOOD_STARS, cstate_matrix, cpchange );
    //debug
    hipMemcpy( outs, couts1, output_size, hipMemcpyDeviceToHost );
    inspect( 'y', outs, NEIGHBORHOOD_STARS, STARS );
  }

  hipFree( couts1 );
  hipFree( couts2 );
  hipFree( crgens );
  hipFree( cstate_matrix );
  hipFree( cpchange );

  return EXIT_SUCCESS;
}
