#include <stdio.h>
#include <stdint.h>
#include "hiprand/hiprand_kernel.h"
#include "stars_config.h"
#include "stars_helpers.h"
#include "stars_helpers_c.h"

int
main()
{
  output_t *outs ;
  output_t *couts1;
  output_t *couts2;
  output_t *cstate_matrix;
  output_t *cstate_matrix_shadow;
  float *cpchange;
  hiprandStateXORWOW_t *crgens;

  unsigned int seed = generate_seed();

  const int output_size = STARS * sizeof( output_t );
  const int rgen_size =   NEIGHBORHOODS * sizeof( hiprandStateXORWOW_t );
  const int state_matrix_size = 2 * NUM_STATES * sizeof( output_t );
  const int pchange_size = NUM_STATES * sizeof( float );

  hipMalloc( (void**)&couts1, output_size );
  hipMalloc( (void**)&couts2, output_size );
  hipMalloc( (void**)&cstate_matrix, state_matrix_size );
  hipMalloc( (void**)&cpchange, pchange_size );
  hipMalloc( (void**)&crgens, rgen_size );


  //TODO: are these being copied to the host correctly?
  hipMemcpy( cstate_matrix, STATE_CHANGES, state_matrix_size, hipMemcpyHostToDevice );
  hipMemcpy( cpchange, P_CHANGE, pchange_size, hipMemcpyHostToDevice );


  init_buf<<<BLOCKS, THREADS_PER_BLOCK>>>( couts1, PROTOSTAR, NEIGHBORHOOD_STARS );
  init_buf<<<BLOCKS, THREADS_PER_BLOCK>>>( couts2, PROTOSTAR, NEIGHBORHOOD_STARS );
  init_rands<<<BLOCKS, THREADS_PER_BLOCK>>>( seed, crgens );


  hipHostAlloc(&cstate_matrix_shadow, state_matrix_size, sizeof(output_t));
  hipMemcpy( cstate_matrix_shadow, cstate_matrix, state_matrix_size, hipMemcpyDeviceToHost );
  inspect( 'S', STATE_CHANGES, 2, state_matrix_size);
  inspect( 's', cstate_matrix_shadow, 2, state_matrix_size);

  hipHostAlloc(&outs, STARS, sizeof(output_t));
  hipMemcpy( outs, couts1, output_size, hipMemcpyDeviceToHost );
  inspect( 'x', outs, NEIGHBORHOOD_STARS, STARS );
  hipMemcpy( outs, couts2, output_size, hipMemcpyDeviceToHost );
  inspect( 'X', outs, NEIGHBORHOOD_STARS, STARS );

  for( int i=0; i< ITERATIONS; i += 2 ){
    iterate_states<<<BLOCKS, THREADS_PER_BLOCK>>>( crgens, couts1, couts2, NEIGHBORHOOD_STARS, cstate_matrix, cpchange );
    iterate_states<<<BLOCKS, THREADS_PER_BLOCK>>>( crgens, couts2, couts1, NEIGHBORHOOD_STARS, cstate_matrix, cpchange );
    //TODO switch to an explicit cuda 3d structure, then compute infections
    hipMemcpy( outs, couts1, output_size, hipMemcpyDeviceToHost );
    inspect( 'y', outs, NEIGHBORHOOD_STARS, STARS );
  }

  hipFree( couts1 );
  hipFree( couts2 );
  hipFree( crgens );
  hipFree( cstate_matrix );
  hipFree( cpchange );

  return EXIT_SUCCESS;
}
