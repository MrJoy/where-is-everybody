#include <stdio.h>
#include <stdint.h>
#include "hiprand/hiprand_kernel.h"
#include "stars_host_device_shared.h"
#include "stars_config.h"
#include "stars_helpers.h"
#include "stars_helpers_c.h"

// 10myr/time step gives 10 Billion year simulation
#define ITERATIONS 1000

int
main()
{
  // output_t *outs ;
  output_t *couts1;
  output_t *couts2;
  hiprandStateXORWOW_t *crgens;
  uint32_t *thread_star_state_counters;
  uint32_t *star_state_counters;
  uint32_t host_star_state_counters[NUM_STATES];

  unsigned int seed = generate_seed();

  const int output_size = STARS * sizeof( output_t );
  const int rgen_size =   THREADS_EVER * sizeof( hiprandStateXORWOW_t );

  hipMalloc( (void**)&couts1, output_size );
  hipMalloc( (void**)&couts2, output_size );
  hipMalloc( (void**)&crgens, rgen_size );
  hipMalloc( (void**)&thread_star_state_counters, NUM_STATES * THREADS_EVER * sizeof(uint32_t) );
  hipMalloc( (void**)&star_state_counters, NUM_STATES * sizeof(uint32_t) );


  init_buf<<<BLOCKS, THREADS_PER_BLOCK>>>(couts1);
  init_buf<<<BLOCKS, THREADS_PER_BLOCK>>>(couts2);
  init_rands<<<BLOCKS, THREADS_PER_BLOCK>>>(seed, crgens);
  init_counters<<<BLOCKS, THREADS_PER_BLOCK>>>(thread_star_state_counters);
  // star_state_counters is re-initialized by each call to sum_states.

  // hipHostAlloc(&outs, STARS, sizeof(output_t));
  hipDeviceSynchronize();

  // for( int i=0; i< ITERATIONS; i += 2 ){
  for( int i=0; i< 100; i += 2 ){
    iterate_states<<<BLOCKS, THREADS_PER_BLOCK>>>(crgens, couts1, couts2);
    hipDeviceSynchronize();
    iterate_states<<<BLOCKS, THREADS_PER_BLOCK>>>(crgens, couts2, couts1);
    hipDeviceSynchronize();
    count_states<<<BLOCKS, THREADS_PER_BLOCK>>>(couts1, thread_star_state_counters);
    hipDeviceSynchronize();
    sum_states<<<1, 1>>>(thread_star_state_counters, star_state_counters);
    hipDeviceSynchronize();
    hipError_t ret = hipMemcpy(host_star_state_counters, star_state_counters, NUM_STATES * sizeof(uint32_t), hipMemcpyDeviceToHost);
    show_counters((int)ret, i, host_star_state_counters);
  }

  hipFree( couts1 );
  hipFree( couts2 );
  hipFree( crgens );
  hipDeviceReset();
  // show_counters(0, host_star_state_counters);

  return EXIT_SUCCESS;
}
