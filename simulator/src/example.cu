/*
 * This program uses the device CURAND API to calculate what
 * proportion of pseudo-random ints have low bit set.
 * It then generates uniform results to calculate how many
 * are greater than .5.
 * It then generates  normal results to calculate how many
 * are within one standard deviation of the mean.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * 64;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void setup_kernel(hiprandStateMRG32k3a *state)
{
    int id = threadIdx.x + blockIdx.x * 64;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(0, id, 0, &state[id]);
}

__global__ void generate_kernel(hiprandState *state,
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    int count = 0;
    unsigned int x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random unsigned ints */
    for(int n = 0; n < 10000; n++) {
        x = hiprand(&localState);
        /* Check if low bit set */
        if(x & 1) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void generate_uniform_kernel(hiprandState *state,
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    unsigned int count = 0;
    float x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random uniforms */
    for(int n = 0; n < 10000; n++) {
        x = hiprand_uniform(&localState);
        /* Check if > .5 */
        if(x > .5) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void generate_normal_kernel(hiprandState *state,
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    unsigned int count = 0;
    float2 x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random normals */
    for(int n = 0; n < 5000; n++) {
        x = hiprand_normal2(&localState);
        /* Check if within one standard deviaton */
        if((x.x > -1.0) && (x.x < 1.0)) {
            count++;
        }
        if((x.y > -1.0) && (x.y < 1.0)) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void generate_kernel(hiprandStateMRG32k3a *state,
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    unsigned int count = 0;
    unsigned int x;
    /* Copy state to local memory for efficiency */
    hiprandStateMRG32k3a localState = state[id];
    /* Generate pseudo-random unsigned ints */
    for(int n = 0; n < 10000; n++) {
        x = hiprand(&localState);
        /* Check if low bit set */
        if(x & 1) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void generate_uniform_kernel(hiprandStateMRG32k3a *state,
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    unsigned int count = 0;
    double x;
    /* Copy state to local memory for efficiency */
    hiprandStateMRG32k3a localState = state[id];
    /* Generate pseudo-random uniforms */
    for(int n = 0; n < 10000; n++) {
        x = hiprand_uniform_double(&localState);
        /* Check if > .5 */
        if(x > .5) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void generate_normal_kernel(hiprandStateMRG32k3a *state,
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    unsigned int count = 0;
    double2 x;
    /* Copy state to local memory for efficiency */
    hiprandStateMRG32k3a localState = state[id];
    /* Generate pseudo-random normals */
    for(int n = 0; n < 5000; n++) {
        x = hiprand_normal2_double(&localState);
        /* Check if within one standard deviaton */
        if((x.x > -1.0) && (x.x < 1.0)) {
            count++;
        }
        if((x.y > -1.0) && (x.y < 1.0)) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

int main(int argc, char *argv[])
{
    int i;
    unsigned int total;
    hiprandState *devStates;
    hiprandStateMRG32k3a *devMRGStates;
    unsigned int *devResults, *hostResults;
    bool useMRG = 0;
    bool doubleSupported = 0;
    int device;
    struct hipDeviceProp_t properties;

    /* check for double precision support */
    CUDA_CALL(hipGetDevice(&device));
    CUDA_CALL(hipGetDeviceProperties(&properties,device));
    if ( properties.major >= 2 || (properties.major == 1 && properties.minor >= 3) ) {
        doubleSupported = 1;
    }

    /* Check for MRG32k3a option (default is XORWOW) */
    if ((argc == 2) && (strcmp(argv[1],"-m") == 0)) {
        useMRG = 1;
        if (!doubleSupported){
            printf("MRG32k3a requires double precision\n");
            printf("^^^^ test WAIVED due to lack of double precision\n");
            return EXIT_SUCCESS;
        }
    }

    /* Allocate space for results on host */
    hostResults = (unsigned int *)calloc(64 * 64, sizeof(int));

    /* Allocate space for results on device */
    CUDA_CALL(hipMalloc((void **)&devResults, 64 * 64 *
              sizeof(unsigned int)));

    /* Set results to 0 */
    CUDA_CALL(hipMemset(devResults, 0, 64 * 64 *
              sizeof(unsigned int)));

    /* Allocate space for prng states on device */
    if (!useMRG) {
        CUDA_CALL(hipMalloc((void **)&devStates, 64 * 64 *
                  sizeof(hiprandState)));
    } else {
        CUDA_CALL(hipMalloc((void **)&devMRGStates, 64 * 64 *
                  sizeof(hiprandStateMRG32k3a)));
    }

    /* Setup prng states */
    if (!useMRG) {
        setup_kernel<<<64, 64>>>(devStates);
    } else {
        setup_kernel<<<64, 64>>>(devMRGStates);
    }

    /* Generate and use pseudo-random  */
    for(i = 0; i < 50; i++) {
        if (!useMRG) {
            generate_kernel<<<64, 64>>>(devStates, devResults);
        } else {
            generate_kernel<<<64, 64>>>(devMRGStates, devResults);
        }
    }

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(hostResults, devResults, 64 * 64 *
        sizeof(unsigned int), hipMemcpyDeviceToHost));

    /* Show result */
    total = 0;
    for(i = 0; i < 64 * 64; i++) {
        total += hostResults[i];
    }
    printf("Fraction with low bit set was %10.13f\n",
        (float)total / (64.0f * 64.0f * 10000.0f * 50.0f));

    /* Set results to 0 */
    CUDA_CALL(hipMemset(devResults, 0, 64 * 64 *
              sizeof(unsigned int)));

    /* Generate and use uniform pseudo-random  */
    for(i = 0; i < 50; i++) {
        if (!useMRG) {
            generate_uniform_kernel<<<64, 64>>>(devStates, devResults);
        } else {
            generate_uniform_kernel<<<64, 64>>>(devMRGStates, devResults);
        }
    }

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(hostResults, devResults, 64 * 64 *
        sizeof(unsigned int), hipMemcpyDeviceToHost));

    /* Show result */
    total = 0;
    for(i = 0; i < 64 * 64; i++) {
        total += hostResults[i];
    }
    printf("Fraction of uniforms > 0.5 was %10.13f\n",
        (float)total / (64.0f * 64.0f * 10000.0f * 50.0f));
    /* Set results to 0 */
    CUDA_CALL(hipMemset(devResults, 0, 64 * 64 *
              sizeof(unsigned int)));

    /* Generate and use uniform pseudo-random  */
    for(i = 0; i < 50; i++) {
        if (!useMRG) {
            generate_normal_kernel<<<64, 64>>>(devStates, devResults);
        } else {
            generate_normal_kernel<<<64, 64>>>(devMRGStates, devResults);
        }
    }

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(hostResults, devResults, 64 * 64 *
        sizeof(unsigned int), hipMemcpyDeviceToHost));

    /* Show result */
    total = 0;
    for(i = 0; i < 64 * 64; i++) {
        total += hostResults[i];
    }
    printf("Fraction of normals within 1 standard deviation was %10.13f\n",
        (float)total / (64.0f * 64.0f * 10000.0f * 50.0f));

    /* Cleanup */
    if (!useMRG) {
        CUDA_CALL(hipFree(devStates));
    } else {
        CUDA_CALL(hipFree(devMRGStates));
    }
    CUDA_CALL(hipFree(devResults));
    free(hostResults);
    printf("^^^^ kernel_example PASSED\n");
    return EXIT_SUCCESS;
}
