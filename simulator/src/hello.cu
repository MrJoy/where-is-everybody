
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hiprand/hiprand_kernel.h"

const int N = 16;
const int blocksize = 16;

__global__
void hello(unsigned int *seeds, unsigned int *outs, int n)
{
  hiprandStateXORWOW_t rgen ;
  hiprand_init(seeds[threadIdx.x], 0,0,&rgen);
  for(int i=0; i<n; ++i){
    outs[threadIdx.x * n + i] = hiprand(&rgen) ;
  }
  //see float curand_uniform
  //see curand__discrete
}

void get_seeds(unsigned int *seeds, int n){
  FILE* randomSource = fopen("/dev/random", "rb");
  unsigned int seed0;
  int recordsRead = fread(&seed0, sizeof(unsigned long), 1, randomSource);
  //assert(recordsRead == n);
  for(int i=0; i<n; ++i){
    seeds[i] = seed0 + i;
  }
  fclose(randomSource);
}

void inspect(unsigned int *array, int n){
  for(int i=0; i< n; ++i){
    printf("%x ", array[i]);
  }
  printf("\n");
}

int main()
{
  unsigned int seeds[N];
  unsigned int outputs[N * N];

  unsigned int *cseeds;
  unsigned int *coutputs;
  const int seed_size = N*sizeof(unsigned int);
  const int output_size = N*N*sizeof(unsigned int);

  get_seeds(&seeds[0], N );
  inspect(seeds, N);

  hipMalloc( (void**)&cseeds, seed_size );
  hipMalloc( (void**)&coutputs, output_size );
  hipMemcpy( cseeds, seeds, seed_size, hipMemcpyHostToDevice );
  hipMemcpy( coutputs, outputs, output_size, hipMemcpyHostToDevice );

  dim3 dimBlock( blocksize, 1 );
  dim3 dimGrid( 1, 1 );
  //TODO, don't pass N, use dim
  hello<<<dimGrid, dimBlock>>>(cseeds, coutputs, N);
  hipMemcpy( outputs, coutputs, output_size, hipMemcpyDeviceToHost );
  hipFree( cseeds );
  hipFree( coutputs );

  for(int i=0; i<N; ++i){
    inspect(outputs + i * N, N );
  }
  return EXIT_SUCCESS;
}
