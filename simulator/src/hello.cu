
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hiprand/hiprand_kernel.h"

const int N = 16;
const int blocksize = 16;

__global__
void
hello(unsigned int *seeds, unsigned int *outs, int n)
{
  hiprandStateXORWOW_t rgen;
  int baseIdx = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init( seeds[baseIdx], 0, 0, &rgen );
  for( int i=0; i<n; ++i ) {
    outs[threadIdx.x * n + i] = hiprand( &rgen );
  }
  // printf("threadIdx.x=%d, blockIdx.x=%d, blockDim.x=%d, seed=%08x\n", threadIdx.x, blockIdx.x, blockDim.x, seeds[threadIdx.x]);
  //see float curand_uniform
  //see curand__discrete
}

void
get_seeds( unsigned int *seeds, int n )
{
  FILE* randomSource = fopen("/dev/random", "rb");
  unsigned int seed0;
  int recordsRead = fread( &seed0, sizeof(unsigned int), 1, randomSource );
  //assert(recordsRead == n);
  for( int i=0; i<n; ++i ) {
    seeds[i] = seed0 + i;
  }
  fclose( randomSource );
}

void
inspect( unsigned int *array, int n )
{
  for( int i=0; i<n; ++i ) {
    printf( "%08x ", array[i] );
  }
  printf( "\n" );
}

int
main()
{
  unsigned int seeds[N];
  unsigned int outputs[N * N];
  for( int j=0; j<N*N; j++ ) {
    outputs[j] = 0;
  }

  unsigned int *cseeds;
  unsigned int *coutputs;
  const int seed_size = N*sizeof( unsigned int );
  const int output_size = N*N*sizeof( unsigned int );

  get_seeds( &seeds[0], N );
  inspect( seeds, N );
  printf( "\n" );

  hipMalloc( (void**)&cseeds, seed_size );
  hipMalloc( (void**)&coutputs, output_size );
  hipMemcpy( cseeds, seeds, seed_size, hipMemcpyHostToDevice );
  hipMemcpy( coutputs, outputs, output_size, hipMemcpyHostToDevice );

  dim3 dimBlock( blocksize, 1 );
  dim3 dimGrid( 1, 1 );
  //TODO, don't pass N, use dim
  hello<<<dimGrid, dimBlock>>>( cseeds, coutputs, N );
  hipMemcpy( outputs, coutputs, output_size, hipMemcpyDeviceToHost );
  hipFree( cseeds );
  hipFree( coutputs );

  for( int i=0; i<N; ++i ) {
    inspect( outputs+i*N, N );
  }
  return EXIT_SUCCESS;
}
