/*
 * This program uses the device CURAND API to calculate what
 * proportion of pseudo-random ints have low bit set.
 * It then generates uniform results to calculate how many
 * are greater than .5.
 * It then generates  normal results to calculate how many
 * are within one standard deviation of the mean.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define THREADS_PER_BLOCK 64
#define BLOCKS 64
#define SIMULTANEOUS_THREADS (THREADS_PER_BLOCK * BLOCKS)
#define RANDOMS_PER_ITERATION 10000
#define KERNEL_ITERATIONS 50


#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void generate_kernel(hiprandState *state,
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    int count = 0;
    unsigned int x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random unsigned ints */
    for(int n = 0; n < RANDOMS_PER_ITERATION; n++) {
        x = hiprand(&localState);
        /* Check if low bit set */
        if(x & 1) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

int main(int argc, char *argv[])
{
    int i;
    unsigned int total;
    hiprandState *devStates;
    unsigned int *devResults, *hostResults;

    /* Allocate space for results on host */
    hostResults = (unsigned int *)calloc(SIMULTANEOUS_THREADS, sizeof(unsigned int));

    /* Allocate space for results on device */
    CUDA_CALL(hipMalloc((void **)&devResults, SIMULTANEOUS_THREADS *
              sizeof(unsigned int)));

    /* Set results to 0 */
    CUDA_CALL(hipMemset(devResults, 0, SIMULTANEOUS_THREADS *
              sizeof(unsigned int)));

    /* Allocate space for prng states on device */
    CUDA_CALL(hipMalloc((void **)&devStates, SIMULTANEOUS_THREADS *
              sizeof(hiprandState)));


    // Set up RNG state objects.
    setup_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(devStates);


    // Generate a ton of random numbers across 50 passes.
    for(i = 0; i < KERNEL_ITERATIONS; i++) {
        generate_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(devStates, devResults);
    }


    // Copy device memory to host.
    CUDA_CALL(hipMemcpy(hostResults, devResults, SIMULTANEOUS_THREADS *
        sizeof(unsigned int), hipMemcpyDeviceToHost));


    // Show result.
    total = 0;
    for(i = 0; i < SIMULTANEOUS_THREADS; i++) {
        total += hostResults[i];
    }
    printf("Fraction with low bit set was %10.13f\n",
        (float)total / (1.0f * SIMULTANEOUS_THREADS * RANDOMS_PER_ITERATION * KERNEL_ITERATIONS));


    /* Cleanup */
    CUDA_CALL(hipFree(devStates));
    CUDA_CALL(hipFree(devResults));
    free(hostResults);
    printf("^^^^ kernel_example PASSED\n");
    return EXIT_SUCCESS;
}
